#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

// Kernel para suma de vectores
__global__ void sumVectors(const float *ptr_v1, const float *ptr_v2,
                           float *ptr_res, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    ptr_res[index] = ptr_v1[index] + ptr_v2[index];
  }
}

int main() {
  const int N = 1000000; // N elementos

  // Inicializar generador de numeros aleatorios con semilla fija
  std::mt19937 gen(322); // Semilla fija
  std::uniform_real_distribution<float> dist(0.0f, 1.0f);

  // Crear vectores en el host
  std::vector<float> v1(N), v2(N), v_res(N);
  for (int i = 0; i < N; ++i) {
    v1[i] = dist(gen);
    v2[i] = dist(gen);
  }

  // Punteros para memoria en el device (GPU)
  float *ptr_v1, *ptr_v2, *ptr_res;

  // Asignar memoria en la GPU
  hipMalloc(&ptr_v1, N * sizeof(float));
  hipMalloc(&ptr_v2, N * sizeof(float));
  hipMalloc(&ptr_res, N * sizeof(float));

  // Copiar datos desde el host al device
  hipMemcpy(ptr_v1, v1.data(), N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(ptr_v2, v2.data(), N * sizeof(float), hipMemcpyHostToDevice);

  // Configurar grid y block
  int blockSize = 256;
  int gridSize = (N + blockSize - 1) / blockSize;

  // Medir tiempo de ejecucion del kernel
  auto start = std::chrono::high_resolution_clock::now();
  sumVectors<<<gridSize, blockSize>>>(ptr_v1, ptr_v2, ptr_res, N);
  hipDeviceSynchronize(); // Esperar a que el kernel termine
  auto end = std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::microseconds>(end - start);

  // Copiar resultado de la GPU al host
  hipMemcpy(v_res.data(), ptr_res, N * sizeof(float), hipMemcpyDeviceToHost);

  // Imprimir tiempo de ejecucion
  std::cout << "Tiempo de ejecucion en GPU: " << duration.count()
            << " microsegundos\n";

  // Verificar algunos resultados
  std::cout << "Primeros 3 resultados:\n";
  for (int i = 0; i < 3 && i < N; ++i) {
    std::cout << v1[i] << " + " << v2[i] << " = " << v_res[i] << "\n";
  }

  std::cout << "Ultimos 3 resultados:\n";
  for (int i = std::max(0, N - 3); i < N; ++i) {
    std::cout << v1[i] << " + " << v2[i] << " = " << v_res[i] << "\n";
  }

  // Liberar memoria del device
  hipFree(ptr_v1);
  hipFree(ptr_v2);
  hipFree(ptr_res);

  return 0;
}
