
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void holaMundo() { printf("Hola Mundo con CUDA\n"); }

int main() {
  holaMundo<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}
